#include "hip/hip_runtime.h"
#ifndef _2DCONVOLUTION_KERNEL_H_
#define _2DCONVOLUTION_KERNEL_H_

#include <stdio.h>
#include "2Dconvolution.h"

// Matrix multiplication kernel thread specification
__global__ void ConvolutionKernelSharedUnroll(Matrix filterd, Matrix Nd, Matrix Pd) {
        __shared__ float fsub[5][5];
        int tx = threadIdx.x; 
        int ty = threadIdx.y; 
        int row = 5 * blockIdx.y + ty;
        int col = 5 * blockIdx.x + tx;
        float value = 0;
        int tempx = 0;
        int tempy = 0;
	int Npitch = Nd.pitch;
	int Nwidth = Nd.width;
	int Nheight = Nd.height;

        fsub[tx][ty] = filterd.elements[ty*filterd.pitch + tx];		// Each thread in the block will import an element from the filter matrix
        __syncthreads();						// Wait until all threads have imported their element.

        for(int i = 0; i < 5; i++) {					// Do the matrix convolution. 
		tempy = row + i - 2;
                tempx = col - 2;
                if(tempx >= 0 && tempx < Nwidth && tempy >= 0 && tempy < Nheight)
                	value += Nd.elements[tempy*Npitch + tempx] * fsub[0][i];
		tempx++;
		if(tempx >= 0 && tempx < Nwidth && tempy >= 0 && tempy < Nheight)
                        value += Nd.elements[tempy*Npitch + tempx] * fsub[1][i];
		tempx++;
		if(tempx >= 0 && tempx < Nwidth && tempy >= 0 && tempy < Nheight)
                        value += Nd.elements[tempy*Npitch + tempx] * fsub[2][i];
		tempx++;
		if(tempx >= 0 && tempx < Nwidth && tempy >= 0 && tempy < Nheight)
                        value += Nd.elements[tempy*Npitch + tempx] * fsub[3][i];
		tempx++;
		if(tempx >= 0 && tempx < Nwidth && tempy >= 0 && tempy < Nheight)
                        value += Nd.elements[tempy*Npitch + tempx] * fsub[4][i];
        }
        if(col < Pd.width && row < Pd.height)
                Pd.elements[row*Pd.pitch + col] = value;
}


// Matrix multiplication kernel thread specification
__global__ void ConvolutionKernelShared(Matrix filterd, Matrix Nd, Matrix Pd) {
	
	__shared__ float fsub[5][5];

	int tx = threadIdx.x;
        int ty = threadIdx.y;
        int row = 5 * blockIdx.y + ty;
        int col = 5 * blockIdx.x + tx;
	float value = 0;
	int tempx = 0;
	int tempy = 0;

	fsub[ty][tx] = filterd.elements[ty*filterd.pitch + tx];
	__syncthreads();

	for(int i = 0; i < 5; i++) {
		for(int j = 0; j < 5; j++) {
			tempx = col + j - 2;
			tempy = row + i - 2;
			if(tempx >= 0 && tempx < Nd.width && tempy >= 0 && tempy < Nd.height)
				value += Nd.elements[tempy*Nd.pitch + tempx] * fsub[i][j];
		}
	}
	if(col < Pd.width && row < Pd.height)
		Pd.elements[row*Pd.pitch + col] = value;
}

// Matrix multiplication kernel thread specification
__global__ void ConvolutionKernel(Matrix filterd, Matrix Nd, Matrix Pd) {
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int row = 5 * blockIdx.y + ty;
        int col = 5 * blockIdx.x + tx;
        float value = 0;
        int tempx = 0;
        int tempy = 0;

        for(int i = 0; i < 5; i++) {
                for(int j = 0; j < 5; j++) {
                        tempx = col + j - 2;
                        tempy = row + i - 2;
                        if(tempx >= 0 && tempx < Nd.width && tempy >= 0 && tempy < Nd.height)
                                value += Nd.elements[tempy*Nd.pitch + tempx] * filterd.elements[i*5 + j];
                }
        }
        if(col < Pd.width && row < Pd.height)
                Pd.elements[row*Pd.pitch + col] = value;
}


#endif // #ifndef _2DCONVOLUTION_KERNEL_H_
