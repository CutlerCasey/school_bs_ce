#include "hip/hip_runtime.h"
/* Matrix multiplication: C = A * B.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <2Dconvolution_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(const Matrix M);
Matrix AllocateMatrix(int height, int width, int init);
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost);
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice);
int ReadFile(Matrix* M, char* file_name);
void WriteFile(Matrix M, char* file_name);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P);


struct timeval begin_time;
struct timeval end_time;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {
	
	double GPU_process_time;
    double CPU_process_time;

    double GPU_start_time;
    double GPU_end_time;

    double CPU_start_time;
    double CPU_end_time;

	Matrix  M;
	Matrix  N;
	Matrix  P;
	
	int N_height = 16; 
	int N_width = 16; 
	
	srand(2012);
	
	if (argc >2 ) {
		N_height = atoi(argv[1]); 
		N_width = atoi(argv[2]); 	
	}
	
	 M  = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE, 1);
	 N  = AllocateMatrix(N_height, N_width, 1);
	 P  = AllocateMatrix(N.height, N.width, 0);

	// M * N on the device
	gettimeofday(&begin_time, NULL);
    ConvolutionOnDevice(M, N, P);
    gettimeofday(&end_time, NULL);
    
    GPU_start_time = (double)begin_time.tv_sec + ((double)begin_time.tv_usec)/1000000;
    GPU_end_time = (double)end_time.tv_sec + (double)end_time.tv_usec/1000000;
    GPU_process_time = GPU_end_time - GPU_start_time;
    
    printf("GPU computation complete\n");    
    printf("GPU Processing time = %f\n\n",GPU_process_time);
    
    // compute the matrix multiplication on the CPU for comparison
    gettimeofday(&begin_time, NULL);
    Matrix reference = AllocateMatrix(P.height, P.width, 0);
    computeGold(reference.elements, M.elements, N.elements, N.height, N.width);
    gettimeofday(&end_time, NULL);
    
    CPU_start_time = (double)begin_time.tv_sec + ((double)begin_time.tv_usec)/1000000;
    CPU_end_time = (double)end_time.tv_sec + (double)end_time.tv_usec/1000000;
    CPU_process_time = CPU_end_time - CPU_start_time;
    
    printf("CPU computation complete\n");
    printf("CPU Processing time = %f\n\n",CPU_process_time);    
    
        
    // in this case check if the result is equivalent to the expected soluion
    CUTBoolean res = cutComparefe(reference.elements, P.elements, P.width * P.height, 0.001f);
    printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");
    
	// Free matrices
    FreeMatrix(&M);
    FreeMatrix(&N);
    FreeMatrix(&P);
	return 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P)
{
    // Load M and N to the device
    Matrix Md = AllocateDeviceMatrix(M);
    CopyToDeviceMatrix(Md, M);
    Matrix Nd = AllocateDeviceMatrix(N);
    CopyToDeviceMatrix(Nd, N);

    // Allocate P on the device
    Matrix Pd = AllocateDeviceMatrix(P);
    CopyToDeviceMatrix(Pd, P); // Clear memory


    
    printf("Nwidth = %d\n",N.width);
    printf("Nheight = %d\n",N.height);		

	    // Setup the execution configuration	
	/* your code starts here */
	
	dim3 DimBlock(5, 5);
	dim3 DimGrid((int)ceil((double)N.width / 5.0), (int)ceil((double)N.height / 5.0));
	ConvolutionKernelSharedUnroll<<< DimGrid, DimBlock >>> (Md, Nd, Pd);
	
	/* your code ends here */ 
	CopyFromDeviceMatrix(P, Pd); 	

    // Free device matrices
    FreeDeviceMatrix(&Md);
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);

}

// Allocate a device matrix of same size as M.
Matrix AllocateDeviceMatrix(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.width * M.height * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Allocate a device matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
//  If init == 2, initialize matrix parameters, but do not allocate memory 
Matrix AllocateMatrix(int height, int width, int init)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;
    M.elements = NULL;
    
    // don't allocate memory on option 2
    if(init == 2)
		return M;
		
	M.elements = (float*) malloc(size*sizeof(float));

	for(unsigned int i = 0; i < M.height * M.width; i++)
	{
		M.elements[i] = (init == 0) ? (0.0f) : (rand() / (float)RAND_MAX);
		if(rand() % 2)
			M.elements[i] = - M.elements[i];
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.width * Mhost.height * sizeof(float);
    Mdevice.height = Mhost.height;
    Mdevice.width = Mhost.width;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, 
					hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice)
{
    int size = Mdevice.width * Mdevice.height * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, 
					hipMemcpyDeviceToHost);
}

// Free a device matrix.
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Free a host Matrix
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}

